////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <fstream>
#include <iostream>
#include <vector>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> 

                        

#include "config.h"     //include config file
#include "kernels.cu"   // include kernel file

#include "hip/hip_runtime_api.h"
using namespace std;

///////////////////////////////////////////////
////////////  CPU FUNCTION TO    //////////////
///////////  READ THE INPUT FILE //////////////
///////////////////////////////////////////////
vector <char> readFile(string inputFile){
    char ch;    
    vector <char> vv;
    fstream fin(inputFile, fstream::in);
    
    while (fin >> noskipws >> ch) {
        if(int(ch)>=MIN_ASCII && int(ch)<=MAX_ASCII && int(ch)!= ASCII_DOLLAR && int(ch) != ASCII_ASTERIK) 
            vv.push_back(ch);
    }

    return vv;
}


///////////////////////////////////////////////
////////////  CPU FUNCTION TO    //////////////
///////////  WRITE HISTOGRAM TO CSV FILE  /////
///////////////////////////////////////////////
void writeResult(char *nameOutputfile, unsigned int *charCount){
    ofstream outputFile(nameOutputfile);

    if(outputFile.is_open()){ 

        for(int i = 0 ; i < NB_CHARS ; i++){
                if(i+MIN_ASCII < ASCII_DOLLAR){
                    outputFile << char(i + MIN_ASCII);
                }
                else if(i+MIN_ASCII >= ASCII_DOLLAR && i+MIN_ASCII<ASCII_ASTERIK - CHAR_DIFF_BET_36_42){

                    outputFile << char(i + MIN_ASCII + CHAR_DIFF_BET_36_42);
                }
                else if(i+MIN_ASCII > ASCII_ASTERIK-CHAR_DIFF_ABOVE_42 && i+MIN_ASCII<UPPERCASE_ALPHABET-CHAR_DIFF_ABOVE_42){
                    outputFile << char(i + MIN_ASCII + CHAR_DIFF_ABOVE_42);
                }
                else{
                    outputFile << char(i + SKIP_UPPERCASE + CHAR_DIFF_ABOVE_42);
                }

                outputFile << " : ";
                outputFile << charCount[i];
                outputFile << "\n";
        }
    }
    outputFile.close();
}

///////////////////////////////////////////////
////////////  CPU FUNCTION TO    //////////////
///////////  GENERATE HISTOGRAM  //////////////
///////////////////////////////////////////////

void genHistogramCPU(vector<char> dict){

    vector<int> count(69, 0);       // used 69 instead of NB_CHARS to remove complexity of ignoreing $ and *

    for(int i=0; i<dict.size(); i++){

        if(int(dict[i]) < UPPERCASE_ALPHABET){
           count[int(dict[i]) - MIN_ASCII] += 1;
        }
        // for uppercase only
        else if(int(dict[i]) >= UPPERCASE_ALPHABET && dict[i] <= 90 ){
            count[int(dict[i]) - MIN_ASCII + 6] += 1;
        }
        // after uppercase characters
        else if(int(dict[i]) > 90){
            count[int(dict[i]) - SKIP_UPPERCASE] += 1;
        }
    }

    ofstream opfile;
    opfile.open("fromCPU.csv");
 
    for(int i = 0 ; i <= 68 ; i++){

        if(i != 4 && i!= 10){
            if(i+MIN_ASCII<UPPERCASE_ALPHABET){
                opfile << char(i + MIN_ASCII);
            }
            else{
                opfile << char(i + SKIP_UPPERCASE);
            }

            opfile << " : ";
            opfile << count[i];
            opfile << "\n";
        }
    }
    opfile.close();
}


///////////////////////////////////////////////
////////////  MAIN PROGRAM  ///////////////////
///////////////////////////////////////////////
int main(int argc, char **argv){
    
    // check if arguments are valid
    if (argc != 5 || strcmp(argv[1], "-i") != 0 || strcmp(argv[3], "-o") != 0) {
        printf("COMMAND FORMAT : ./template -i <inputText.txt> -o <outputHisto.csv> \n");
        return -1;
    }

    cout<<"BLOCKS "<<NB_BLOCKS<<" AND THREADS "<<NB_THREADS<<endl;

    // get input and output file names    
    char *inputFile = argv[2]; 
    char *outputFile = argv[4];

    // read the input text file
    vector<char> charDict = readFile(inputFile);
    

    //                                 SECTION CPU
    ///////////////////////////////////////////////////////////////////////////////////
    
    StopWatchInterface *cpu_timer = 0;
    sdkCreateTimer(&cpu_timer);
    sdkStartTimer(&cpu_timer);
    
    // call the function
    genHistogramCPU(charDict);
    
    // end and del timer
    sdkStopTimer(&cpu_timer);
    printf("Processing time of CPU(in ms) : %f \n", sdkGetTimerValue(&cpu_timer));
    sdkDeleteTimer(&cpu_timer);


    //                                 SECTION GPU
    ///////////////////////////////////////////////////////////////////////////////////


    unsigned int COUNTCHAR = charDict.size();
    cout<<"Size of countchar is "<<COUNTCHAR<<endl;

    // For HOST
    char *allChars;             // to contain all the characters read
    unsigned int *charCount;    // total number of characters in the file
    

    // Allocation of host copy
    allChars = (char*)malloc(COUNTCHAR*sizeof(char));
    charCount = (unsigned int*)malloc(NB_CHARS*sizeof(unsigned int));
    
    // copy the characters from vector to array
    copy(charDict.begin(), charDict.end(), allChars);

 

    // allocation of device copies
    char *devAllChars;                  // device copy of all characters from file
    unsigned int *devCharCount;         // to contain the final histogram
    unsigned int *devBlockCharCount;    // for while using shared memory, this will contain the data for each block

    
    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // allocation of device memory
    hipMalloc((void**)&devAllChars, COUNTCHAR*sizeof(char));
    hipMalloc((void**)&devCharCount, NB_CHARS*sizeof(unsigned int));

    // copy the host data to device
    hipMemcpy(devAllChars, allChars, COUNTCHAR*sizeof(char), hipMemcpyHostToDevice);
    hipMemset(devCharCount, 0, NB_CHARS*sizeof(unsigned int));
    


    // Call the kernel to generate the histogram 
    genHistogram <<< NB_BLOCKS, NB_THREADS >>>(devAllChars, devCharCount, COUNTCHAR);
    


    // copy device result back to host copy
    hipMemcpy(charCount, devCharCount, NB_CHARS*sizeof(unsigned int), hipMemcpyDeviceToHost);
    
    // end and del timer
    sdkStopTimer(&timer);
    printf("GPU Processing time (in ms)[naive method] : %f \n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);


    // write the results to output file
    writeResult(outputFile, charCount);


    // free device momory
    hipFree(devCharCount);
    hipFree(devAllChars); 


    //////    USING SHARED MEMORY
    ////////////////////////////////////////////////////////////////

    StopWatchInterface *timer2 = 0;
    sdkCreateTimer(&timer2);
    sdkStartTimer(&timer2);

    // Allocation of device copy
    hipMalloc((void**)&devAllChars, COUNTCHAR*sizeof(char));
    hipMalloc((void**)&devCharCount, NB_CHARS*sizeof(unsigned int));
    hipMalloc((void**)&devBlockCharCount, NB_CHARS*NB_BLOCKS*sizeof(unsigned int));

    // copy the host data to device
    hipMemcpy(devAllChars, allChars, COUNTCHAR*sizeof(char), hipMemcpyHostToDevice);
    hipMemset(devCharCount, 0, NB_CHARS*sizeof(unsigned int));
    hipMemset(devBlockCharCount, 0, NB_CHARS*NB_BLOCKS*sizeof(unsigned int));

    // hipProfilerStart();

    // call the kernel to create histogram for each block
    genHistogramShared <<< NB_BLOCKS, NB_THREADS >>>(devAllChars, devBlockCharCount, COUNTCHAR);
    

    // hipProfilerStop();
    
    // combine the data of all blocks to produce one result containing the histogram
    combineBlockData<<< 1, NB_CHARS >>>(devBlockCharCount, devCharCount, NB_CHARS*NB_BLOCKS);

    // copy device result back to host copy
    hipMemcpy(charCount, devCharCount, NB_CHARS*sizeof(unsigned int), hipMemcpyDeviceToHost);
    
    // end and del timer
    sdkStopTimer(&timer2);
    printf("GPU Processing time (in ms)[using shared memory] : %f \n", sdkGetTimerValue(&timer2));
    sdkDeleteTimer(&timer2);

    // write the results to output file
    writeResult(outputFile, charCount);


    // free device momory
    hipFree(devCharCount);
    hipFree(devAllChars); 
    hipFree(devBlockCharCount);

    // free host memory
    free(allChars);
    free(charCount);

}